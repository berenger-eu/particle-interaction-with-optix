#include "hip/hip_runtime.h"
//
// Copyright (c) Inria 2024
//

#include <optix.h>

#include "optixTriangle.h"
#include <cuda/helpers.h>

#include <sutil/vec_math.h>

extern "C" {
__constant__ ParamsLJ params;
}


static __forceinline__ __device__ void trace(
        OptixTraversableHandle handle,
        float3                 ray_origin,
        float3                 ray_direction,
        float                  tmin,
        float                  tmax,
        float3                 partPos,
        float                  c,
        float*                 energy,
        unsigned int           ray_idx
        )
{
    unsigned int p0, p1, p2, p3, p4, p5;
    p0 = __float_as_uint( partPos.x );
    p1 = __float_as_uint( partPos.y );
    p2 = __float_as_uint( partPos.z );
    p3 = __float_as_uint( *energy );
    p4 = __float_as_uint( c );
    p5 = ray_idx;

    optixTrace(
            handle,
            ray_origin,
            ray_direction,
            tmin,
            tmax,
            0.0f,                // rayTime
            OptixVisibilityMask( 1 ),
            OPTIX_RAY_FLAG_NONE,
            0,                   // SBT offset
            0,                   // SBT stride
            0,                   // missSBTIndex
            p0, p1, p2, p3, p4, p5);
    
    (*energy) += __uint_as_float( p3 );
}
static __forceinline__ __device__ unsigned int getPayloadRayidx()
{
    return optixGetPayload_5();
}

static __forceinline__ __device__ float getPayloadC()
{
    return __uint_as_float( optixGetPayload_4() );
}

static __forceinline__ __device__ void setPayloadEnergy( float p )
{
    optixSetPayload_3( __float_as_uint( p ) );
}

static __forceinline__ __device__ float getPayloadEnergy()
{
    return __uint_as_float( optixGetPayload_3() );
}

static __forceinline__ __device__ float3 getPayloadPartPos()
{
    float3 point;
    point.x = __uint_as_float( optixGetPayload_0() );
    point.y = __uint_as_float( optixGetPayload_1() );
    point.z = __uint_as_float( optixGetPayload_2() );
    return point;
}

extern "C" __global__ void __raygen__rg()
{
    const uint3 idx = optixGetLaunchIndex();
    // const uint3 dim = optixGetLaunchDimensions();
    const int point_index = idx.x;

    const RayGenDataLJ* rtData = (RayGenDataLJ*)optixGetSbtDataPointer();
    float3 point;
    point.x = params.points[point_index];
    point.y = params.points[point_index + params.leading_dim];
    point.z = params.points[point_index + params.leading_dim*2];
    const float c = params.c;
    const float half_c = params.c/2;
    const float half_ray = (params.c/2);

    // Coordinates are:
    //  1 --- 5
    //  |\   |\
    //  | \  | \
    //  0 -3- 4  7
    //   \ |  \ |   
    //    \|   \|
    //     2 --- 6
    // First triangle is 0,1,3
    // Second triangle is 0,2,3
    // Third triangle is 4,5,7
    // Fourth triangle is 4,6,7

    float3 origin;
    float3 direction;
    {
        float ycoef = (idx.y & 1) ? -1.0f : 1.0f;
        float zcoef = (idx.y & 2) ? -1.0f : 1.0f;

        origin = make_float3(point.x - half_ray,
                             point.y + half_c * ycoef,
                             point.z + half_c * zcoef);
        direction = make_float3(1, 0, 0);
    }

    float payload_energy = 0;
    const float feps = 1.19209290e-07F;
    const float tmin = feps;
    const float tmax = (2 * half_ray) + (2 * half_ray) * 0.00001;
    trace( params.handle,
            origin,
            direction,
            tmin,
            tmax,
            point,
            c,
            &payload_energy ,
            idx.y);
    
    atomicAdd(&params.energy[point_index], payload_energy);
}


extern "C" __global__ void __miss__ms()
{
}


static __forceinline__ __device__  float  distance(const float3& p1, const float3& p2) {
    return sqrt((p2.x - p1.x)*(p2.x - p1.x) + (p2.y - p1.y)*(p2.y - p1.y) + (p2.z - p1.z)*(p2.z - p1.z));
}

// Function to calculate the Lennard-Jones potential between two particles
static __forceinline__ __device__  float  lennardJonesPotential(const float3 p1, 
                                                                const float3 p2, 
                                                                const float dist_p1_p2,
                                                                const float epsilon, 
                                                                const float sigma) {
    const float r = dist_p1_p2;//distance(p1, p2);
    const float sigma_d_r = sigma / r;
    const float r6 = (sigma_d_r*sigma_d_r)*(sigma_d_r*sigma_d_r)*(sigma_d_r*sigma_d_r);
    const float r12 = r6 * r6;
    const float result = float(4) * epsilon * (r12 - r6);
    return result;
}


extern "C" __global__ void __anyhit__ch()
{
    const unsigned int           prim_idx    = optixGetPrimitiveIndex();
    const OptixTraversableHandle gas         = optixGetGASTraversableHandle();
    const unsigned int           sbtGASIndex = optixGetSbtGASIndex();

    float3 vertices[3];
    optixGetTriangleVertexData( gas, prim_idx, sbtGASIndex, 0.f, vertices );

    float3 q;
    q.y = (max(vertices[0].y,max(vertices[1].y, vertices[2].y)) + min(vertices[0].y,min(vertices[1].y, vertices[2].y)))/2;
    q.z = (max(vertices[0].z,max(vertices[1].z, vertices[2].z)) + min(vertices[0].z,min(vertices[1].z, vertices[2].z)))/2;

    const float c = getPayloadC();
    if((prim_idx % 4) < 2){
        q.x = vertices[0].x + c/2;
    }
    else{
        q.x = vertices[0].x - c/2;
    }

    const float3 point = getPayloadPartPos();

    const float dist_p1_p2 = distance(point, q);
    if(dist_p1_p2 < c && dist_p1_p2 > 0.0001){
        // const float3 ray_orig = optixGetWorldRayOrigin();
        // const float3 ray_dir  = optixGetWorldRayDirection();

        const unsigned int ray_idx = getPayloadRayidx();
        const bool is_ray_for_compute = (point.y != q.y && point.z != q.z) ||
                                        ((point.z < q.z && ray_idx == 0) || (point.z > q.z && ray_idx == 2)) ||
                                        ((point.y < q.y && ray_idx == 0) || (point.y > q.y && ray_idx == 1)) ||
                                        ray_idx == 0;// y and z are same

        if(is_ray_for_compute){
            // const float3 hit_position = optixGetWorldRayOrigin() + optixGetRayTmax() * optixGetWorldRayDirection();
            const float epsilon = 1.0f;
            const float sigma = 1.0f;
            const float energy = lennardJonesPotential(point, q, dist_p1_p2,
                                                       epsilon, sigma);

            setPayloadEnergy( getPayloadEnergy() + energy );
        }
    }

    // Backface hit not used.
    // float  t_hit2 = __uint_as_float( optixGetAttribute_0() ); 
    // float3 world_raypos = ray_orig + t_hit * ray_dir;
    // float3 obj_raypos   = optixTransformPointFromWorldToObjectSpace( world_raypos );
    // float3 obj_normal   = ( obj_raypos - make_float3( q ) ) / q.w;
    // float3 world_normal = normalize( optixTransformNormalFromObjectToWorldSpace( obj_normal ) );
    // optixTerminateRay();
    optixIgnoreIntersection();
}
